#include "hip/hip_runtime.h"
/*
Christopher Ginac

image.cpp
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <iostream>
#include "Image.cuh"
#include <cmath>
using namespace std;
const int ntpb = 1024;

__global__ void negate(int* a, int* b, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n) {
		a[idx] = -(b[idx]) + 255;
	}
}

__global__ void verticalReflect(int* a, int* b, int sz, int n, int m) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x = idx / m;
	int y = idx % m;
	if (idx < sz) {
		//a[idx] = b[x * m + (m - y)];
		a[x * m + (m - y)] = b[idx];
	}
}

__global__ void horizontalReflect(int* a, int* b,int sz, int n, int m) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x = idx / m;
	int y = idx % m;
	if (idx < sz) {
		a[(n - x) * m + y] = b[idx];
	}
}

Image::Image()
/* Creates an Image 0x0 */
{
	N = 0;
	M = 0;
	Q = 0;

	pixelVal = NULL;
}

Image::Image(int numRows, int numCols, int grayLevels)
/* Creates an Image of numRows x numCols and creates the arrays for it*/
{

	N = numRows;
	M = numCols;
	Q = grayLevels;

	pixelVal = new int[N * M];
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
			pixelVal[i * M + j] = 0;
	}
}

Image::~Image()
/*destroy image*/
{
	N = 0;
	M = 0;
	Q = 0;
	delete pixelVal;
}

Image::Image(const Image& oldImage)
/*copies oldImage into new Image object*/
{
	N = oldImage.N;
	M = oldImage.M;
	Q = oldImage.Q;

	pixelVal = new int[N * M];
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
			pixelVal[i * M + j] = oldImage.pixelVal[i * M + j];
	}
}

void Image::operator=(const Image& oldImage)
/*copies oldImage into whatever you = it to*/
{
	N = oldImage.N;
	M = oldImage.M;
	Q = oldImage.Q;

	pixelVal = new int[N * M];
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
			pixelVal[i * M + j] = oldImage.pixelVal[i * M + j];
	}
}

void Image::setImageInfo(int numRows, int numCols, int maxVal)
/*sets the number of rows, columns and graylevels*/
{
	N = numRows;
	M = numCols;
	Q = maxVal;
}

void Image::getImageInfo(int &numRows, int &numCols, int &maxVal)
/*returns the number of rows, columns and gray levels*/
{
	numRows = N;
	numCols = M;
	maxVal = Q;
}

int Image::getPixelVal(int row, int col)
/*returns the gray value of a specific pixel*/
{
	return pixelVal[row * M + col];
}


void Image::setPixelVal(int row, int col, int value)
/*sets the gray value of a specific pixel*/
{
	pixelVal[row * M + col] = value;
}

bool Image::inBounds(int row, int col)
/*checks to see if a pixel is within the image, returns true or false*/
{
	if (row >= N || row < 0 || col >= M || col < 0)
		return false;
	//else
	return true;
}

void Image::getSubImage(int upperLeftRow, int upperLeftCol, int lowerRightRow,
	int lowerRightCol, Image& oldImage)
	/*Pulls a sub image out of oldImage based on users values, and then stores it
	in oldImage*/
{
	int width, height;

	width = lowerRightCol - upperLeftCol;
	height = lowerRightRow - upperLeftRow;

	Image tempImage(height, width, Q);

	for (int i = upperLeftRow; i < lowerRightRow; i++)
	{
		for (int j = upperLeftCol; j < lowerRightCol; j++)
			tempImage.pixelVal[(i - upperLeftRow) * height + j - upperLeftCol] = oldImage.pixelVal[i * oldImage.M + j];
	}

	oldImage = tempImage;
}

int Image::meanGray()
/*returns the mean gray levels of the Image*/
{
	int totalGray = 0;

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
			totalGray += pixelVal[i * M + j];
	}

	int cells = M * N;

	return (totalGray / cells);
}

void Image::enlargeImage(int value, Image& oldImage)
/*enlarges Image and stores it in tempImage, resizes oldImage and stores the
larger image in oldImage*/
{
	int rows, cols, gray;
	int pixel;
	int enlargeRow, enlargeCol;

	rows = oldImage.N * value;
	cols = oldImage.M * value;
	gray = oldImage.Q;

	Image tempImage(rows, cols, gray);

	for (int i = 0; i < oldImage.N; i++)
	{
		for (int j = 0; j < oldImage.M; j++)
		{
			pixel = oldImage.pixelVal[i * oldImage.M + j];
			enlargeRow = i * value;
			enlargeCol = j * value;
			for (int c = enlargeRow; c < (enlargeRow + value); c++)
			{
				for (int d = enlargeCol; d < (enlargeCol + value); d++)
				{
					tempImage.pixelVal[c * cols + d] = pixel;
				}
			}
		}
	}

	oldImage = tempImage;
}

void Image::shrinkImage(int value, Image& oldImage)
/*Shrinks image as storing it in tempImage, resizes oldImage, and stores it in
oldImage*/
{
	int rows, cols, gray;

	rows = oldImage.N / value;
	cols = oldImage.M / value;
	gray = oldImage.Q;

	Image tempImage(rows, cols, gray);

	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
			tempImage.pixelVal[i * cols + j] = oldImage.pixelVal[(i * value) * cols + j * value];
	}
	oldImage = tempImage;
}

void Image::reflectImage(bool flag, Image& oldImage)
/*Reflects the Image based on users input*/
{
	int rows = oldImage.N;
	int cols = oldImage.M;
	Image tempImage(oldImage);
	/*
	if (flag == true) //horizontal reflection
	{
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
				tempImage.pixelVal[(rows - (i + 1)) * cols + j] = oldImage.pixelVal[i * cols + j];
		}
	}
	else //vertical reflection
	{
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
				tempImage.pixelVal[i * cols + cols - (j + 1)] = oldImage.pixelVal[i * cols + j];
		}
	}
	*/
	int* d_temp = nullptr;
	int* d_img = nullptr;
	int size = rows * cols;
	int nblocks = size / ntpb;
	hipMalloc((void**)&d_temp, size * sizeof(int));
	hipMalloc((void**)&d_img, size * sizeof(int));
	hipMemcpy(d_temp, tempImage.pixelVal, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_img, oldImage.pixelVal, size * sizeof(int), hipMemcpyHostToDevice);
	if (flag) {
		horizontalReflect<<<nblocks, ntpb>>>(d_temp, d_img, size, rows, cols);
	}
	else {
		verticalReflect<<<nblocks, ntpb>>>(d_temp, d_img, size, rows, cols);
	}
	hipDeviceSynchronize();
	hipMemcpy(tempImage.pixelVal, d_temp, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_temp);
	hipFree(d_img);
	
	oldImage = tempImage;
}

void Image::translateImage(int value, Image& oldImage)
/*translates image down and right based on user value*/
{
	int rows = oldImage.N;
	int cols = oldImage.M;
	int gray = oldImage.Q;
	Image tempImage(N, M, Q);

	for (int i = 0; i < (rows - value); i++)
	{
		for (int j = 0; j < (cols - value); j++)
			tempImage.pixelVal[(i + value) * cols + j + value] = oldImage.pixelVal[i * cols + j];
	}

	oldImage = tempImage;
}

void Image::rotateImage(int theta, Image& oldImage)
/*based on users input and rotates it around the center of the image.*/
{
	int r0, c0;
	int r1, c1;
	int rows, cols;
	rows = oldImage.N;
	cols = oldImage.M;
	Image tempImage(rows, cols, oldImage.Q);

	float rads = (theta * 3.14159265) / 180.0;

	r0 = rows / 2;
	c0 = cols / 2;

	for (int r = 0; r < rows; r++)
	{
		for (int c = 0; c < cols; c++)
		{
			r1 = (int)(r0 + ((r - r0) * cos(rads)) - ((c - c0) * sin(rads)));
			c1 = (int)(c0 + ((r - r0) * sin(rads)) + ((c - c0) * cos(rads)));

			if (inBounds(r1, c1))
			{
				tempImage.pixelVal[r1 * cols + c1] = oldImage.pixelVal[r * cols + c];
			}
		}
	}

	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			if (tempImage.pixelVal[i * cols + j] == 0)
				tempImage.pixelVal[i * cols + j] = tempImage.pixelVal[i * cols + j + 1];
		}
	}
	oldImage = tempImage;
}

Image Image::operator+(const Image &oldImage)
/*adds images together, half one image, half the other*/
{
	Image tempImage(oldImage);

	int rows, cols;
	rows = oldImage.N;
	cols = oldImage.M;

	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
			tempImage.pixelVal[i * cols + j] = (pixelVal[i * cols + j] + oldImage.pixelVal[i * cols + j]) / 2;
	}

	return tempImage;
}

Image Image::operator-(const Image& oldImage)
/*subtracts images from each other*/
{
	Image tempImage(oldImage);

	int rows, cols;
	rows = oldImage.N;
	cols = oldImage.M;
	int tempGray = 0;

	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{

			tempGray = abs(pixelVal[i * cols + j] - oldImage.pixelVal[i * cols + j]);
			if (tempGray < 35)// accounts for sensor flux
				tempGray = 0;
			tempImage.pixelVal[i * cols + j] = tempGray;
		}

	}

	return tempImage;
}

void Image::negateImage(Image& oldImage)
/*negates image*/
{
	Image tempImage(N, M, Q);

	/*for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
			tempImage.pixelVal[i * cols + j] = -(pixelVal[i * cols + j]) + 255;
	}*/

	int* d_temp = nullptr;
	int* d_img = nullptr;
	int size = N * M;
	int nblocks = size / ntpb;
	hipMalloc((void**)&d_temp, size * sizeof(int));
	hipMalloc((void**)&d_img, size * sizeof(int));
	hipMemcpy(d_temp, tempImage.pixelVal, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_img, pixelVal, size * sizeof(int), hipMemcpyHostToDevice);
	negate<<<nblocks, ntpb>>>(d_temp, d_img, size);
	hipDeviceSynchronize();
	hipMemcpy(tempImage.pixelVal, d_temp, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_temp);
	hipFree(d_img);

	oldImage = tempImage;
}
