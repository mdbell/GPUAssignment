#include "hip/hip_runtime.h"
/*
Christopher Ginac

image.cpp
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <iostream>
#include "Image.cuh"
#include <cmath>
using namespace std;
const int ntpb = 1024;

__global__ void enlarge(int* a, int* b, int sz, int scale, int cols, int scols) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x = idx / scols;
	int y = idx % scols;
	if (idx < sz) {
		a[idx] = b[(x / scale) * cols + (y / scale)];
		//a[idx] = 0xFFFFFFFF;
	}
}

__global__ void negate(int* a, int* b, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n) {
		a[idx] = -(b[idx]) + 255;
	}
}

__global__ void verticalReflect(int* a, int* b, int sz, int n, int m) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x = idx / m;
	int y = idx % m;
	if (idx < sz) {
		//a[idx] = b[x * m + (m - y)];
		a[x * m + (m - y)] = b[idx];
	}
}

__global__ void horizontalReflect(int* a, int* b, int sz, int n, int m) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x = idx / m;
	int y = idx % m;
	if (idx < sz) {
		a[(n - x) * m + y] = b[idx];
	}
}

Image::Image()
/* Creates an Image 0x0 */
{
	N = 0;
	M = 0;
	Q = 0;

	pixelVal = NULL;
}

Image::Image(int numRows, int numCols, int grayLevels)
/* Creates an Image of numRows x numCols and creates the arrays for it*/
{

	N = numRows;
	M = numCols;
	Q = grayLevels;
	hipMalloc((void**)&pixelVal, N * M * sizeof(int));
	hipMemset(pixelVal,0,N * M * sizeof(int));
	/*	pixelVal = new int[N * M];
	for (int i = 0; i < N; i++)
	{
	for (int j = 0; j < M; j++)
	pixelVal[i * M + j] = 0;
	}
	*/
}

Image::~Image()
/*destroy image*/
{
	if (pixelVal) {
		hipFree(pixelVal);
	}
	//delete pixelVal;
}

Image::Image(const Image& oldImage)
/*copies oldImage into new Image object*/
{
	N = oldImage.N;
	M = oldImage.M;
	Q = oldImage.Q;
	int sz = M * N * sizeof(int);
	hipMalloc((void**)&pixelVal, sz);
	hipMemcpy(pixelVal, oldImage.pixelVal, sz, hipMemcpyDeviceToDevice);
}

void Image::operator=(const Image& oldImage)
/*copies oldImage into whatever you = it to*/
{
	N = oldImage.N;
	M = oldImage.M;
	Q = oldImage.Q;

	if (pixelVal) {
		hipFree(pixelVal);
	}

	int sz = M * N * sizeof(int);
	hipMalloc((void**)&pixelVal, sz);
	hipMemcpy(pixelVal, oldImage.pixelVal, sz, hipMemcpyDeviceToDevice);
}

void Image::setImageInfo(int numRows, int numCols, int maxVal)
/*sets the number of rows, columns and graylevels*/
{
	N = numRows;
	M = numCols;
	Q = maxVal;
}

void Image::getImageInfo(int &numRows, int &numCols, int &maxVal)
/*returns the number of rows, columns and gray levels*/
{
	numRows = N;
	numCols = M;
	maxVal = Q;
}

int Image::getPixelVal(int row, int col)
/*returns the gray value of a specific pixel*/
{
	int i = 0;
	int* idx = pixelVal + (row * M + col);
	hipMemcpy(&i, idx, sizeof(int), hipMemcpyDeviceToHost);
	return i;
}


void Image::setPixelVal(int row, int col, int value)
/*sets the gray value of a specific pixel*/
{
	int* idx = pixelVal + (row * M + col);
	hipMemcpy(idx, &value, sizeof(int), hipMemcpyHostToDevice);
//	pixelVal[row * M + col] = value;
}

void Image::getPixels(int row, int col, int sz, int* out) {
	hipMemcpy(out, pixelVal + (row * M + col), sz * sizeof(int), hipMemcpyDeviceToHost);
}

void Image::setPixels(int row, int col, int sz, int* in) {
		hipMemcpy(pixelVal + (row * M + col), in, sz * sizeof(int), hipMemcpyHostToDevice);
}

void Image::enlargeImage(int value, Image& oldImage)
/*enlarges Image and stores it in tempImage, resizes oldImage and stores the
larger image in oldImage*/
{
	int rows, cols, gray;
	int pixel;
	int enlargeRow, enlargeCol;

	rows = oldImage.N * value;
	cols = oldImage.M * value;
	gray = oldImage.Q;

	Image tempImage(rows, cols, gray);


	int r = oldImage.N;
	int c = oldImage.M;

	int* d_temp = tempImage.pixelVal;
	int* d_img = oldImage.pixelVal;
	int size = rows * cols;
	int nblocks = size / ntpb;

	//hipMalloc((void**)&d_temp, size * sizeof(int));
	//hipMalloc((void**)&d_img, size * sizeof(int));

	//hipMemcpy(d_temp, tempImage.pixelVal, size * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(d_img, oldImage.pixelVal, (r * c) * sizeof(int), hipMemcpyHostToDevice);

	enlarge << <nblocks, ntpb >> >(d_temp, d_img, size, value, c, cols);

	hipDeviceSynchronize();

	//set the image's data
	//hipMemcpy(tempImage.pixelVal, d_temp, size * sizeof(int), hipMemcpyDeviceToHost);

	//free device mem
	//hipFree(d_temp);
	//hipFree(d_img);
	/*

	for (int i = 0; i < oldImage.N; i++)
	{
	for (int j = 0; j < oldImage.M; j++)
	{
	pixel = oldImage.pixelVal[i * oldImage.M + j];
	enlargeRow = i * value;
	enlargeCol = j * value;
	for (int c = enlargeRow; c < (enlargeRow + value); c++)
	{
	for (int d = enlargeCol; d < (enlargeCol + value); d++)
	{
	tempImage.pixelVal[c * cols + d] = pixel;
	}
	}
	}
	}
	/**/
	oldImage = tempImage;
}

void Image::reflectImage(bool flag, Image& oldImage)
/*Reflects the Image based on users input*/
{
	int rows = oldImage.N;
	int cols = oldImage.M;
	Image tempImage(oldImage);

	int* d_temp = tempImage.pixelVal;
	int* d_img = oldImage.pixelVal;
	
	int size = rows * cols;
	int nblocks = size / ntpb;
	//hipMalloc((void**)&d_temp, size * sizeof(int));
	//hipMalloc((void**)&d_img, size * sizeof(int));
	//hipMemcpy(d_temp, tempImage.pixelVal, size * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(d_img, oldImage.pixelVal, size * sizeof(int), hipMemcpyHostToDevice);
	if (flag) {
		horizontalReflect << <nblocks, ntpb >> >(d_temp, d_img, size, rows, cols);
	}
	else {
		verticalReflect << <nblocks, ntpb >> >(d_temp, d_img, size, rows, cols);
	}
	hipDeviceSynchronize();
	//hipMemcpy(tempImage.pixelVal, d_temp, size * sizeof(int), hipMemcpyDeviceToHost);
	//hipFree(d_temp);
	//hipFree(d_img);

	oldImage = tempImage;
}

void Image::negateImage(Image& oldImage)
/*negates image*/
{
	Image tempImage(N, M, Q);

	/*for (int i = 0; i < rows; i++)
	{
	for (int j = 0; j < cols; j++)
	tempImage.pixelVal[i * cols + j] = -(pixelVal[i * cols + j]) + 255;
	}*/

	int* d_temp = tempImage.pixelVal;
	int* d_img = pixelVal;
	int size = N * M;
	int nblocks = size / ntpb;
	
	//hipMalloc((void**)&d_temp, size * sizeof(int));
	//hipMalloc((void**)&d_img, size * sizeof(int));
	//hipMemcpy(d_temp, tempImage.pixelVal, size * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(d_img, pixelVal, size * sizeof(int), hipMemcpyHostToDevice);

	negate << <nblocks, ntpb >> >(d_temp, d_img, size);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));

	hipDeviceSynchronize();
	//hipMemcpy(tempImage.pixelVal, d_temp, size * sizeof(int), hipMemcpyDeviceToHost);
	//hipFree(d_temp);
	//hipFree(d_img);

	oldImage = tempImage;
}
